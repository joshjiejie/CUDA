#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>
#include <random>
#include <cstdio>

__global__
void gpu_transpose(
    float * a,
    int  dim1,
    int  dim2,
    int  dim3,
    int  dim4
) {
      int c = threadIdx.x;
      int h = blockIdx.x;
      int w = blockIdx.y;
      float tmp;
      int start = (h<w) ? 0 : dim1/2;
      int end = (h<w) ? dim1/2 : dim1;
   
      if(h!=w) {
        for (int n = start; n < end; n++) {
          tmp = a [n*dim2*dim3*dim4 + h*dim3*dim4 + w*dim4 + dim4/2 + c];
          a [n*dim2*dim3*dim4 + h*dim3*dim4 + w*dim4 + dim4/2 + c] = a [n*dim2*dim3*dim4 + w*dim3*dim4 + h*dim4 + dim4/2 + c];
          a [n*dim2*dim3*dim4 + w*dim3*dim4 + h*dim4 + dim4/2 + c] = tmp;
        }
      }
}

void cpu_transpose(
    float * a,
    float * b,
    int  dim1,
    int  dim2,
    int  dim3,
    int  dim4
) {
    for(int i=0; i<dim1; i++)
      for(int j=0; j<dim2; j++)
        for(int k=0; k<dim3; k++)
          for(int w=0; w<dim4; w++)
              if(w<dim4/2)
                b[i*dim2*dim3*dim4 + j*dim3*dim4 + k*dim4 + w] = a[i*dim2*dim3*dim4 + j*dim3*dim4 + k*dim4 + w];
              else
                b[i*dim2*dim3*dim4 + j*dim3*dim4 + k*dim4 + w] = a[i*dim2*dim3*dim4 + k*dim3*dim4 + j*dim4 + w];
}

void random_fill(float * data, int num_elems) {
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<> dis(-10.0, 10.0);
    for (int i = 0; i < num_elems; i++) {
        data[i] = dis(gen);
    }
}

int main(int argc, char ** argv) {
    constexpr int N = 4;
    constexpr int H = 32;
    constexpr int W = 32;
    constexpr int C = 1024;
    constexpr int C_by_2 = C/2;

    float * x, * cpu_result;
    float * x_device, * gpu_result;
    constexpr int x_size = sizeof(float) * N * H * W * C;

    hipStream_t stream;
    hipStreamCreate(&stream);

    hipHostMalloc(&x,  x_size);
    hipHostMalloc(&cpu_result, x_size);
    hipHostMalloc(&gpu_result, x_size);
  
    random_fill(x, N * H * W * C);

    hipMalloc(&x_device, x_size);
    hipMemcpyAsync(x_device, x, x_size, hipMemcpyHostToDevice, stream);

    dim3 block(C_by_2);
    dim3 grid (H, W);
              
    gpu_transpose<<<grid, block, 0, stream>>>(x_device, N, H, W, C);

    hipMemcpyAsync(gpu_result, x_device, x_size, hipMemcpyDeviceToHost, stream);
    hipStreamSynchronize(stream);

    int mismatch = 0;

    cpu_transpose(x, cpu_result, N, H, W, C);

    for(int i=0; i<N; i++)
      for(int j=0; j<H; j++)
        for(int k=0; k<W; k++)
          for(int w=0; w<C; w++)
            if (gpu_result[i*H*W*C+ j*W*C + k*C + w] != cpu_result[i*H*W*C+ j*W*C + k*C + w]) {
				mismatch++;
			}

    printf("mismatch: %d\n", mismatch);
    

    // Clean up
    hipFree(x_device);
    hipHostFree(x);
    hipHostFree(gpu_result);
    hipHostFree(cpu_result);

    hipStreamDestroy(stream);

    return 0;
}